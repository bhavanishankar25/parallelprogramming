#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include <cstdio> 
#include <iostream> 

__global__ void vecAdd( double *a,   double *b,  double *c, int n)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int bs = blockDim.x;
    for (int k = id; k < n; k += bs) {
        c[k] = a[k] + b[k];
    }
    
}
 
int main( int argc, char* argv[] )
{
    int n = 100000000;
 
    
    double *h_a;
    double *h_b;
    double *h_c;
 
    double *d_a;
    double *d_b;
    double *d_c;
 
    size_t bytes = n*sizeof(double);
 
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = i*(double) rand() + 0.00001534784187571354923459*(double) rand();
        h_b[i] = i*(double) rand() + 0.00001534784187571354923459*(double) rand();
    }
 
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize =500;
    int gridSize =1;

 
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    //for(i=0; i<n; i++)
    //printf("%f\n, ", h_c[i]);
 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}