#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include <stdio.h> 
#include <iostream>

static const int arraySize = 100000000;
static const int blockSize = 500;
__global__ void dotproduct(double *a, double *b,double *out) {
    int idx = threadIdx.x;
    double sum = 0;
    for (int i = idx; i < arraySize; i += blockSize)
        sum += a[i]*b[i];
    __shared__ double r[blockSize];
    r[idx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (idx<size)
            r[idx] += r[idx+size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}


int main(){
    

    double *h_a;
    double *h_b;
    double *h_result;

    double *d_a;
    double *d_b;
    double *d_result;

    

    size_t bytes = arraySize*sizeof(double);

    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_result = (double*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_result, bytes);

    int i;
    for( i = 0; i < arraySize; i++ ) {
        h_a[i] = i*(double) rand() + 0.00001534784187571354923459*(double) rand();
        h_b[i] = i*(double) rand() + 0.00001534784187571354923459*(double) rand();
    }

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize =500;
    int gridSize =1;

    dotproduct<<< gridSize, blockSize >>>(d_a,d_b,d_result);

    hipMemcpy( h_result, d_result, bytes, hipMemcpyDeviceToHost );


    hipFree(d_a);
    hipFree(d_result);
 
    free(h_a);
    free(h_result);


    return 0;
}

